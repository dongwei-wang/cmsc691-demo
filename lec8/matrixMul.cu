
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matrixMul(float *A, float *B, float *C, int width)
{
	int column = ( blockDim.x * blockIdx.x ) + threadIdx.x;
	int row    = ( blockDim.y * blockIdx.y ) + threadIdx.y;

	if (row < width && column < width)
	{
		float sum = 0;

		for(int k = 0; k < width; k++)
			sum += A[row * width + k] + B[k * width + column];

		C[row*width + column] = sum;
	}
}

void MatrixMultiplicationHost(float *A, float *B, float *C, int width)
{
	for (int i = 0; i < width; i++)
		for (int j = 0; j < width; j++)
		{
			float sum = 0;

			for (int k = 0; k < width; k++)
				sum += A[i * width + k] + B[k * width + j];

			C[i * width + j] = sum;
		}
}

int main(int argc, char* argv[])
{
	int matrixSize = 1024; // square matrix matrixSize * matrixSize
	int numElements = matrixSize * matrixSize;

	// Allocate host memory
	float *h_A = (float *)malloc(numElements * sizeof(float));
	float *h_B = (float *)malloc(numElements * sizeof(float));
	float *h_C = (float *)malloc(numElements * sizeof(float));
	float *h_C_CPUres = (float *)malloc(numElements * sizeof(float));

	// Initialize the host input matrixs
	for (int i = 0; i < numElements; ++i)
	{
		h_A[i] = rand()/(float)RAND_MAX;
		h_B[i] = rand()/(float)RAND_MAX;
	}

	// Allocate the device input matrix A
	float *d_A, *d_B, *d_C;

	hipMalloc(&d_A, numElements * sizeof(float));
	hipMalloc(&d_B, numElements * sizeof(float));
	hipMalloc(&d_C, numElements * sizeof(float));

	// Copy the host input matrixs A and B in host memory to the device input matrixs in
	hipMemcpy(d_A, h_A, numElements * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, numElements * sizeof(float), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;

	int threadsPerBlockDim = 16;
	int gridDimSize = (matrixSize + threadsPerBlockDim - 1) / threadsPerBlockDim;

	dim3 blockSize(threadsPerBlockDim, threadsPerBlockDim);
	dim3 gridSize (gridDimSize, gridDimSize);

	printf("CUDA kernel launch with %dx%d blocks of %dx%d threads\n", gridDimSize, gridDimSize, threadsPerBlockDim, threadsPerBlockDim);

	hipEventRecord(start);

	matrixMul<<<gridSize, blockSize>>>(d_A, d_B, d_C, matrixSize);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("GPU time to multiple matrixes %f ms\n", milliseconds);

	// Copy the device result matrix in device memory to the host result matrix
	hipMemcpy(h_C, d_C, numElements * sizeof(float), hipMemcpyDeviceToHost);

	hipError_t hipError_t = hipGetLastError();

	if(hipError_t != hipSuccess)
	{
		fprintf(stderr, "cudaGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
		exit(EXIT_FAILURE);
	}

	// Compute CPU time
	hipEventRecord(start);

	MatrixMultiplicationHost(h_A, h_B, h_C_CPUres, matrixSize);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("CPU time to sum the matrixes %f ms\n", milliseconds);

	// Verify that the result matrix is correct
	for (int i = 0; i < numElements; i++)
		if (fabs(h_C[i] - h_C_CPUres[i]) > 1e-5)
		{
			fprintf(stderr, "Result verification failed at element %d, %f vs %f!\n", i, h_C[i], h_C_CPUres[i]);
			exit(EXIT_FAILURE);
		}

	printf("Multiplication of the matrixes was OK\n");

	// Free device global memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// Free host memory
	free(h_A);
	free(h_B);
	free(h_C);
	free(h_C_CPUres);

	return 0;
}
