
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(float *A, float *B, float *C, int numElements)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < numElements)
        C[tid] = A[tid] + B[tid];
}

int main(int argc, char* argv[])
{
    int numElements = 50000;

    // Allocate host memory
    float *h_A = (float *)malloc(numElements * sizeof(float));
    float *h_B = (float *)malloc(numElements * sizeof(float));
    float *h_C = (float *)malloc(numElements * sizeof(float));

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vector A
    float *d_A, *d_B, *d_C;

    hipMalloc(&d_A, numElements * sizeof(float));
    hipMalloc(&d_B, numElements * sizeof(float));
    hipMalloc(&d_C, numElements * sizeof(float));

    // Copy the host input vectors A and B in host memory to the device input vectors in
    hipMemcpy(d_A, h_A, numElements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, numElements * sizeof(float), hipMemcpyHostToDevice);

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // Copy the device result vector in device memory to the host result vector
    hipMemcpy(h_C, d_C, numElements * sizeof(float), hipMemcpyDeviceToHost);
    hipError_t hipError_t = hipGetLastError();

    if(hipError_t != hipSuccess)
    {
        fprintf(stderr, "cudaGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; i++)
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }

    printf("Sum of the vectors was OK\n");

    // Free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
