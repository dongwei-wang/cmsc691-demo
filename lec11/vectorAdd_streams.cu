
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(float *A, float *B, float *C, int stream, int numberElementsPerStream)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int idx = stream * numberElementsPerStream + tid;

    if (tid < numberElementsPerStream)
        C[idx] = A[idx] + B[idx];
}

int main(int argc, char* argv[])
{
    int numElements = pow(2,20); // 2^20 approximately 1M elements
    int numStreams = 4;

    // Allocate host memory
    float *h_A, *h_B, *h_C;

    hipHostMalloc(&h_A, numElements * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_B, numElements * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_C, numElements * sizeof(float), hipHostMallocDefault);

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vector A
    float *d_A, *d_B, *d_C;

    hipMalloc(&d_A, numElements * sizeof(float));
    hipMalloc(&d_B, numElements * sizeof(float));
    hipMalloc(&d_C, numElements * sizeof(float));

    hipStream_t *streams = (hipStream_t*) malloc (numStreams * sizeof(hipStream_t));

    for (int i = 0; i < numStreams; i++)
        hipStreamCreate(&streams[i]);

    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;

	hipEventRecord(start);

	int threadsPerBlock = 256;
	int numberElementsPerStream = (numElements + numStreams - 1) / numStreams;
	int blocksPerGrid = (numberElementsPerStream + threadsPerBlock - 1) / threadsPerBlock;

	for (int i = 0; i < numStreams; i++)
	{
		// Copy the host input vectors A and B in host memory to the device input vectors in
		hipMemcpyAsync(&d_A[i*numberElementsPerStream],
						&h_A[i*numberElementsPerStream],
						numberElementsPerStream * sizeof(float),
						hipMemcpyHostToDevice,
						streams[i]);
		hipMemcpyAsync(&d_B[i*numberElementsPerStream],
						&h_B[i*numberElementsPerStream],
						numberElementsPerStream * sizeof(float),
						hipMemcpyHostToDevice,
						streams[i]);

		vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, streams[i]>>>(d_A, d_B, d_C, i, numberElementsPerStream);

		// Copy the device result vector in device memory to the host result vector
		hipMemcpyAsync(&h_C[i*numberElementsPerStream],
						&d_C[i*numberElementsPerStream],
						numberElementsPerStream * sizeof(float),
						hipMemcpyDeviceToHost,
						streams[i]);
	}

	hipDeviceSynchronize();

    hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("GPU time %d streams %f ms\n", numStreams, milliseconds);

    hipError_t hipError_t = hipGetLastError();

    if(hipError_t != hipSuccess)
    {
        fprintf(stderr, "cudaGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; i++)
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }

    printf("Sum of the vectors was OK\n");

    // Free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);

    return 0;
}

