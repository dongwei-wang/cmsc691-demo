
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(float *A, float *B, float *C, int numElements)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < numElements)
        C[tid] = A[tid] + B[tid];
}

int main(int argc, char* argv[])
{
    int numElements = pow(2,20); // 2^20 approximately 1M elements

    // Allocate host memory
    float *h_A, *h_B, *h_C;

    hipHostMalloc(&h_A, numElements * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_B, numElements * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_C, numElements * sizeof(float), hipHostMallocDefault);

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vector A
    float *d_A, *d_B, *d_C;

    hipMalloc(&d_A, numElements * sizeof(float));
    hipMalloc(&d_B, numElements * sizeof(float));
    hipMalloc(&d_C, numElements * sizeof(float));

    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;

	hipEventRecord(start);

    // Copy the host input vectors A and B in host memory to the device input vectors in
    hipMemcpy(d_A, h_A, numElements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, numElements * sizeof(float), hipMemcpyHostToDevice);

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // Copy the device result vector in device memory to the host result vector
    hipMemcpy(h_C, d_C, numElements * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("GPU time single stream %f ms\n", milliseconds);

    hipError_t hipError_t = hipGetLastError();

    if(hipError_t != hipSuccess)
    {
        fprintf(stderr, "cudaGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; i++)
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }

    printf("Sum of the vectors was OK\n");

    // Free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);

    return 0;
}
