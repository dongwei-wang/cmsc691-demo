
#include <hip/hip_runtime.h>
#include <stdio.h>

const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;

// Check errors and print GB/s
void postprocess(const float *ref, const float *res, int n, float ms)
{
    bool passed = true;
    for (int i = 0; i < n; i++)
        if (res[i] != ref[i]) {
          printf("%d %f %f\n", i, res[i], ref[i]);
          printf("%25s\n", "*** FAILED ***");
          passed = false;
          break;
     }

    if (passed)
        printf("%20.2f\n", 2 * n * sizeof(float) * 1e-6 / ms );
}

// naive transpose
// Simplest transpose; doesn't use shared memory.
// Global memory reads are coalesced but writes are not.
__global__ void transposeNaive(float *odata, const float *idata)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
        odata[x*width + (y+j)] = idata[(y+j)*width + x];
}

// coalesced transpose
// Uses shared memory to achieve coalesing in both reads and writes
// Tile width == #banks causes shared memory bank conflicts.
__global__ void transposeCoalesced(float *odata, const float *idata)
{
    __shared__ float tile[TILE_DIM][TILE_DIM];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

    __syncthreads();

    x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}


// No bank-conflict transpose
// Same as transposeCoalesced except the first tile dimension is padded
// to avoid shared memory bank conflicts.
__global__ void transposeNoBankConflicts(float *odata, const float *idata)
{
    __shared__ float tile[TILE_DIM][TILE_DIM+1];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

    __syncthreads();

    x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}

int main(int argc, char **argv)
{
    const int nx = 1024;
    const int ny = 1024;
    const int mem_size = nx*ny*sizeof(float);

    dim3 dimGrid(nx/TILE_DIM, ny/TILE_DIM, 1);
    dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);

    int devId = 0;
    if (argc > 1) devId = atoi(argv[1]);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, devId);
    printf("\nDevice : %s\n", prop.name);
    printf("Matrix size: %d %d, Block size: %d %d, Tile size: %d %d\n", nx, ny, TILE_DIM, BLOCK_ROWS, TILE_DIM, TILE_DIM);
    printf("dimGrid: %d %d %d. dimBlock: %d %d %d\n", dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);

    hipSetDevice(devId);

    float *h_idata = (float*)malloc(mem_size);
    float *h_cdata = (float*)malloc(mem_size);
    float *h_tdata = (float*)malloc(mem_size);
    float *gold    = (float*)malloc(mem_size);

    float *d_idata, *d_cdata, *d_tdata;
    hipMalloc(&d_idata, mem_size);
    hipMalloc(&d_cdata, mem_size);
    hipMalloc(&d_tdata, mem_size);

    // check parameters and calculate execution configuration
    if (nx % TILE_DIM || ny % TILE_DIM) {
        printf("nx and ny must be a multiple of TILE_DIM\n");
        goto error_exit;
    }

    if (TILE_DIM % BLOCK_ROWS) {
        printf("TILE_DIM must be a multiple of BLOCK_ROWS\n");
        goto error_exit;
        }

    // host
    for (int j = 0; j < ny; j++)
        for (int i = 0; i < nx; i++)
          h_idata[j*nx + i] = j*nx + i;

    // correct result for error checking
    for (int j = 0; j < ny; j++)
        for (int i = 0; i < nx; i++)
          gold[j*nx + i] = h_idata[i*nx + j];

    // device
    hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice);

    // events for timing
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    float ms;

    // ------------
    // time kernels
    // ------------
    printf("%25s%25s\n", "Routine", "Bandwidth (GB/s)");

    // --------------
    // transposeNaive
    // --------------
    printf("%25s", "naive transpose");
    hipMemset(d_tdata, 0, mem_size);
    // warmup
    transposeNaive<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
    hipEventRecord(startEvent, 0);
    transposeNaive<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost);
    postprocess(gold, h_tdata, nx * ny, ms);

    // ------------------
    // transposeCoalesced
    // ------------------
    printf("%25s", "coalesced transpose");
    hipMemset(d_tdata, 0, mem_size);
    // warmup
    transposeCoalesced<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
    hipEventRecord(startEvent, 0);
    transposeCoalesced<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost);
    postprocess(gold, h_tdata, nx * ny, ms);

    // ------------------------
    // transposeNoBankConflicts
    // ------------------------
    printf("%25s", "conflict-free transpose");
    hipMemset(d_tdata, 0, mem_size);
    // warmup
    transposeNoBankConflicts<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
    hipEventRecord(startEvent, 0);
    transposeNoBankConflicts<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost);
    postprocess(gold, h_tdata, nx * ny, ms);

    error_exit:
    // cleanup
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    hipFree(d_tdata);
    hipFree(d_cdata);
    hipFree(d_idata);
    free(h_idata);
    free(h_tdata);
    free(h_cdata);
    free(gold);
}
