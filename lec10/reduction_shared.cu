
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void reduce_shared(int *result, int *array, int numElements)
{
    __shared__ int sharedMemory[256];

    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    sharedMemory[threadIdx.x] = (tid < numElements) ? array[tid] : 0;

    __syncthreads();

    // do reduction in shared memory
    for (int s = blockDim.x/2; s > 0; s >>= 1)
	{
		if (threadIdx.x < s)
			sharedMemory[threadIdx.x] += sharedMemory[threadIdx.x + s];

		__syncthreads();
	}

    // write result for this block to global memory
    if (threadIdx.x == 0)
        atomicAdd(result, sharedMemory[0]);
}

int main(int argc, char* argv[])
{
    int numElements = 1e6;

    // Allocate host memory
    int *h_array  = (int *)malloc(numElements * sizeof(int));
    int *h_result = (int *)malloc(sizeof(int));

    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;

    // Initialize the host input vectors
    for (int i = 0; i < numElements; i++)
        h_array[i] = (i+1);

    // Allocate the device input vector
    int *d_array, *d_result;
    hipMalloc(&d_array, numElements * sizeof(int));
    hipMalloc(&d_result, sizeof(int));

    // Copy the host input vector
    hipMemcpy(d_array, h_array, numElements * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_result, 0, sizeof(int));

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    printf("%d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    hipEventRecord(start);

    reduce_shared<<<blocksPerGrid, threadsPerBlock>>>(d_result, d_array, numElements);

    hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("GPU time %f ms\n", milliseconds);

    // Copy the result
    hipMemcpy(h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    hipError_t hipError_t = hipGetLastError();

    if(hipError_t != hipSuccess)
    {
        fprintf(stderr, "cudaGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
        exit(EXIT_FAILURE);
    }

    hipEventRecord(start);

    int CPU_result = 0;

    for (int i = 0; i < numElements; i++)
    	CPU_result += h_array[i];

    hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("CPU time %f ms\n", milliseconds);

    printf("GPU result %d, CPU result %d, %s!\n", *h_result, CPU_result, *h_result == CPU_result ? "CORRECT" : "ERROR" );

    // Free device global memory
    hipFree(d_array);
    hipFree(d_result);

    // Free host memory
    free(h_result);

    return 0;
}

