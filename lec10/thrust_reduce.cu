#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <numeric>
#include <stdio.h>

int main(int argc, char* argv[])
{
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;

    // generate 16M random numbers on the host
    thrust::host_vector<int> h_vec(1 << 24);

    thrust::generate(h_vec.begin(), h_vec.end(), rand);

    // transfer data to the device
    thrust::device_vector<int> d_vec = h_vec;

    hipEventRecord(start);

    // sort data on the device
    int result_GPU = thrust::reduce(d_vec.begin(), d_vec.end());

    hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("GPU time %f ms\n", milliseconds);

	// transfer data back to host
    thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

	hipEventRecord(start);

	int result_CPU = std::accumulate(h_vec.begin(), h_vec.end(), 0);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("CPU time %f ms\n", milliseconds);

	printf("GPU result %d\nCPU result %d\n", result_GPU, result_CPU);

    return 0;
}
